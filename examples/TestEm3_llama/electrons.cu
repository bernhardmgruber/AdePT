#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2021 CERN
// SPDX-License-Identifier: Apache-2.0

#include "TestEm3.cuh"

#include <AdePT/BVHNavigator.h>
#include <fieldPropagatorConstBz.h>

#include <CopCore/PhysicalConstants.h>

#define NOFLUCTUATION

#include <G4HepEmElectronManager.hh>
#include <G4HepEmElectronTrack.hh>
#include <G4HepEmElectronInteractionBrem.hh>
#include <G4HepEmElectronInteractionIoni.hh>
#include <G4HepEmElectronInteractionUMSC.hh>
#include <G4HepEmPositronInteractionAnnihilation.hh>
// Pull in implementation.
#include <G4HepEmRunUtils.icc>
#include <G4HepEmInteractionUtils.icc>
#include <G4HepEmElectronManager.icc>
#include <G4HepEmElectronInteractionBrem.icc>
#include <G4HepEmElectronInteractionIoni.icc>
#include <G4HepEmElectronInteractionUMSC.icc>
#include <G4HepEmPositronInteractionAnnihilation.icc>

// Compute the physics and geometry step limit, transport the electrons while
// applying the continuous effects and maybe a discrete process that could
// generate secondaries.
template <bool IsElectron>
static __device__ __forceinline__ void TransportElectrons(View electrons, const adept::MParray *active,
                                                          Secondaries &secondaries, adept::MParray *activeQueue,
                                                          GlobalScoring *globalScoring,
                                                          ScoringPerVolume *scoringPerVolume)
{
  constexpr int Charge  = IsElectron ? -1 : 1;
  constexpr double Mass = copcore::units::kElectronMassC2;
  fieldPropagatorConstBz fieldPropagatorBz(BzFieldValue);

  int activeSize = active->size();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < activeSize; i += blockDim.x * gridDim.x) {
    const int slot       = (*active)[i];
    auto &&currentTrack  = electrons[slot];
    const int volumeID   = currentTrack(NavState{}).Top()->id();
    const int theMCIndex = MCIndex[volumeID];

    auto survive = [&] { activeQueue->push_back(slot); };

    // Init a track with the needed data to call into G4HepEm.
    G4HepEmElectronTrack elTrack;
    G4HepEmTrack *theTrack = elTrack.GetTrack();
    theTrack->SetEKin(currentTrack(Energy{}));
    theTrack->SetMCIndex(theMCIndex);
    theTrack->SetOnBoundary(currentTrack(NavState{}).IsOnBoundary());
    theTrack->SetCharge(Charge);
    G4HepEmMSCTrackData *mscData = elTrack.GetMSCTrackData();
    mscData->fIsFirstStep        = currentTrack(InitialRange{}) < 0;
    mscData->fInitialRange       = currentTrack(InitialRange{});
    mscData->fDynamicRangeFactor = currentTrack(DynamicRangeFactor{});
    mscData->fTlimitMin          = currentTrack(TlimitMin{});

    // Prepare a branched RNG state while threads are synchronized. Even if not
    // used, this provides a fresh round of random numbers and reduces thread
    // divergence because the RNG state doesn't need to be advanced later.
    auto rngRef = makeRngRef(currentTrack(RngState{})); // reference to RNG in memory
    auto newRNG = rngRef.BranchNoAdvance();

    // Compute safety, needed for MSC step limit.
    double safety = 0;
    if (!currentTrack(NavState{}).IsOnBoundary()) {
      safety = BVHNavigator::ComputeSafety(currentTrack(Pos{}), currentTrack(NavState{}));
    }
    theTrack->SetSafety(safety);

    RanluxppDoubleEngineRef rnge(&rngRef);

    // Sample the `number-of-interaction-left` and put it into the track.
    for (int ip = 0; ip < 3; ++ip) {
      double &numIALeft = currentTrack(NumIALeft{})[ip];
      if (numIALeft <= 0) {
        numIALeft = -std::log(rngRef.Rndm());
      }
      theTrack->SetNumIALeft(numIALeft, ip);
    }

    // Call G4HepEm to compute the physics step limit.
    G4HepEmElectronManager::HowFar(&g4HepEmData, &g4HepEmPars, &elTrack, &rnge);

    // Remember MSC values for the next step(s).
    currentTrack(InitialRange{})       = mscData->fInitialRange;
    currentTrack(DynamicRangeFactor{}) = mscData->fDynamicRangeFactor;
    currentTrack(TlimitMin{})          = mscData->fTlimitMin;

    // Get result into variables.
    double geometricalStepLengthFromPhysics = theTrack->GetGStepLength();
    // The phyiscal step length is the amount that the particle experiences
    // which might be longer than the geometrical step length due to MSC. As
    // long as we call PerformContinuous in the same kernel we don't need to
    // care, but we need to make this available when splitting the operations.
    // double physicalStepLength = elTrack.GetPStepLength();
    int winnerProcessIndex = theTrack->GetWinnerProcessIndex();
    // Leave the range and MFP inside the G4HepEmTrack. If we split kernels, we
    // also need to carry them over!

    // Check if there's a volume boundary in between.
    bool propagated = true;
    double geometryStepLength;
    vecgeom::NavStateIndex nextState;
    if (BzFieldValue != 0) {
      geometryStepLength = fieldPropagatorBz.ComputeStepAndNextVolume<BVHNavigator>(
          currentTrack(Energy{}), Mass, Charge, geometricalStepLengthFromPhysics, currentTrack(Pos{}),
          currentTrack(Dir{}), currentTrack(NavState{}), nextState, propagated, safety);
    } else {
      geometryStepLength =
          BVHNavigator::ComputeStepAndNextVolume(currentTrack(Pos{}), currentTrack(Dir{}),
                                                 geometricalStepLengthFromPhysics, currentTrack(NavState{}), nextState);
      currentTrack(Pos{}) += geometryStepLength * currentTrack(Dir{});
    }

    // Set boundary state in navState so the next step and secondaries get the
    // correct information (currentTrack.navState = nextState only if relocated
    // in case of a boundary; see below)
    currentTrack(NavState{}).SetBoundaryState(nextState.IsOnBoundary());

    // Propagate information from geometrical step to MSC.
    theTrack->SetDirection(currentTrack(Dir{}).x(), currentTrack(Dir{}).y(), currentTrack(Dir{}).z());
    theTrack->SetGStepLength(geometryStepLength);
    theTrack->SetOnBoundary(nextState.IsOnBoundary());

    // Apply continuous effects.
    bool stopped = G4HepEmElectronManager::PerformContinuous(&g4HepEmData, &g4HepEmPars, &elTrack, &rnge);

    // Collect the direction change and displacement by MSC.
    const double *direction = theTrack->GetDirection();
    currentTrack(Dir{}).Set(direction[0], direction[1], direction[2]);
    if (!nextState.IsOnBoundary()) {
      const double *mscDisplacement = mscData->GetDisplacement();
      vecgeom::Vector3D<Precision> displacement(mscDisplacement[0], mscDisplacement[1], mscDisplacement[2]);
      const double dLength2            = displacement.Length2();
      constexpr double kGeomMinLength  = 5 * copcore::units::nm;          // 0.05 [nm]
      constexpr double kGeomMinLength2 = kGeomMinLength * kGeomMinLength; // (0.05 [nm])^2
      if (dLength2 > kGeomMinLength2) {
        const double dispR = std::sqrt(dLength2);
        // Estimate safety by subtracting the geometrical step length.
        safety -= geometryStepLength;
        constexpr double sFact = 0.99;
        double reducedSafety   = sFact * safety;

        // Apply displacement, depending on how close we are to a boundary.
        // 1a. Far away from geometry boundary:
        if (reducedSafety > 0.0 && dispR <= reducedSafety) {
          currentTrack(Pos{}) += displacement;
        } else {
          // Recompute safety.
          safety        = BVHNavigator::ComputeSafety(currentTrack(Pos{}), currentTrack(NavState{}));
          reducedSafety = sFact * safety;

          // 1b. Far away from geometry boundary:
          if (reducedSafety > 0.0 && dispR <= reducedSafety) {
            currentTrack(Pos{}) += displacement;
            // 2. Push to boundary:
          } else if (reducedSafety > kGeomMinLength) {
            currentTrack(Pos{}) += displacement * (reducedSafety / dispR);
          }
          // 3. Very small safety: do nothing.
        }
      }
    }

    // Collect the charged step length (might be changed by MSC).
    atomicAdd(&globalScoring->chargedSteps, 1);
    atomicAdd(&scoringPerVolume->chargedTrackLength[volumeID], elTrack.GetPStepLength());

    // Collect the changes in energy and deposit.
    currentTrack(Energy{}) = theTrack->GetEKin();
    double energyDeposit   = theTrack->GetEnergyDeposit();
    atomicAdd(&globalScoring->energyDeposit, energyDeposit);
    atomicAdd(&scoringPerVolume->energyDeposit[volumeID], energyDeposit);

    // Save the `number-of-interaction-left` in our track.
    for (int ip = 0; ip < 3; ++ip) {
      double numIALeft              = theTrack->GetNumIALeft(ip);
      currentTrack(NumIALeft{})[ip] = numIALeft;
    }

    if (stopped) {
      if (!IsElectron) {
        // Annihilate the stopped positron into two gammas heading to opposite
        // directions (isotropic).
        auto &&gamma1 = secondaries.gammas.NextTrack();
        auto &&gamma2 = secondaries.gammas.NextTrack();
        atomicAdd(&globalScoring->numGammas, 2);

        const double cost = 2 * rngRef.Rndm() - 1;
        const double sint = sqrt(1 - cost * cost);
        const double phi  = k2Pi * rngRef.Rndm();
        double sinPhi, cosPhi;
        sincos(phi, &sinPhi, &cosPhi);

        InitAsSecondary(gamma1, /*parent=*/currentTrack);
        newRNG.Advance();
        storeRng(gamma1(RngState{}), newRNG);
        gamma1(Energy{}) = copcore::units::kElectronMassC2;
        gamma1(Dir{}).Set(sint * cosPhi, sint * sinPhi, cost);

        InitAsSecondary(gamma2, /*parent=*/currentTrack);
        // Reuse the RNG state of the dying track.
        gamma2(RngState{}) = currentTrack(RngState{});
        gamma2(Energy{})   = copcore::units::kElectronMassC2;
        gamma2(Dir{})      = -gamma1(Dir{});
      }
      // Particles are killed by not enqueuing them into the new activeQueue.
      continue;
    }

    if (nextState.IsOnBoundary()) {
      // For now, just count that we hit something.
      atomicAdd(&globalScoring->hits, 1);

      // Kill the particle if it left the world.
      if (nextState.Top() != nullptr) {
        BVHNavigator::RelocateToNextVolume(currentTrack(Pos{}), currentTrack(Dir{}), nextState);

        // Move to the next boundary.
        currentTrack(NavState{}) = nextState;
        survive();
      }
      continue;
    } else if (!propagated) {
      // Did not yet reach the interaction point due to error in the magnetic
      // field propagation. Try again next time.
      survive();
      continue;
    } else if (winnerProcessIndex < 0) {
      // No discrete process, move on.
      survive();
      continue;
    }

    // Reset number of interaction left for the winner discrete process.
    // (Will be resampled in the next iteration.)
    currentTrack(NumIALeft{})[winnerProcessIndex] = -1.0;

    // Check if a delta interaction happens instead of the real discrete process.
    if (G4HepEmElectronManager::CheckDelta(&g4HepEmData, theTrack, rngRef.Rndm())) {
      // A delta interaction happened, move on.
      survive();
      continue;
    }

    // Perform the discrete interaction, make sure the branched RNG state is
    // ready to be used.
    newRNG.Advance();
    // Also advance the current RNG state to provide a fresh round of random
    // numbers after MSC used up a fair share for sampling the displacement.
    rngRef.Advance();

    const double energy   = currentTrack(Energy{});
    const double theElCut = g4HepEmData.fTheMatCutData->fMatCutData[theMCIndex].fSecElProdCutE;

    switch (winnerProcessIndex) {
    case 0: {
      // Invoke ionization (for e-/e+):
      double deltaEkin = (IsElectron) ? G4HepEmElectronInteractionIoni::SampleETransferMoller(theElCut, energy, &rnge)
                                      : G4HepEmElectronInteractionIoni::SampleETransferBhabha(theElCut, energy, &rnge);

      double dirPrimary[] = {currentTrack(Dir{}).x(), currentTrack(Dir{}).y(), currentTrack(Dir{}).z()};
      double dirSecondary[3];
      G4HepEmElectronInteractionIoni::SampleDirections(energy, deltaEkin, dirSecondary, dirPrimary, &rnge);

      auto &&secondary = secondaries.electrons.NextTrack();
      atomicAdd(&globalScoring->numElectrons, 1);

      InitAsSecondary(secondary, /*parent=*/currentTrack);
      storeRng(secondary(RngState{}), newRNG);
      secondary(Energy{}) = deltaEkin;
      secondary(Dir{}).Set(dirSecondary[0], dirSecondary[1], dirSecondary[2]);

      currentTrack(Energy{}) = energy - deltaEkin;
      currentTrack(Dir{}).Set(dirPrimary[0], dirPrimary[1], dirPrimary[2]);
      survive();
      break;
    }
    case 1: {
      // Invoke model for Bremsstrahlung: either SB- or Rel-Brem.
      double logEnergy = std::log(energy);
      double deltaEkin = energy < g4HepEmPars.fElectronBremModelLim
                             ? G4HepEmElectronInteractionBrem::SampleETransferSB(&g4HepEmData, energy, logEnergy,
                                                                                 theMCIndex, &rnge, IsElectron)
                             : G4HepEmElectronInteractionBrem::SampleETransferRB(&g4HepEmData, energy, logEnergy,
                                                                                 theMCIndex, &rnge, IsElectron);

      double dirPrimary[] = {currentTrack(Dir{}).x(), currentTrack(Dir{}).y(), currentTrack(Dir{}).z()};
      double dirSecondary[3];
      G4HepEmElectronInteractionBrem::SampleDirections(energy, deltaEkin, dirSecondary, dirPrimary, &rnge);

      auto &&gamma = secondaries.gammas.NextTrack();
      atomicAdd(&globalScoring->numGammas, 1);

      InitAsSecondary(gamma, /*parent=*/currentTrack);
      storeRng(gamma(RngState{}), newRNG);
      gamma(Energy{}) = deltaEkin;
      gamma(Dir{}).Set(dirSecondary[0], dirSecondary[1], dirSecondary[2]);

      currentTrack(Energy{}) = energy - deltaEkin;
      currentTrack(Dir{}).Set(dirPrimary[0], dirPrimary[1], dirPrimary[2]);
      survive();
      break;
    }
    case 2: {
      // Invoke annihilation (in-flight) for e+
      double dirPrimary[] = {currentTrack(Dir{}).x(), currentTrack(Dir{}).y(), currentTrack(Dir{}).z()};
      double theGamma1Ekin, theGamma2Ekin;
      double theGamma1Dir[3], theGamma2Dir[3];
      G4HepEmPositronInteractionAnnihilation::SampleEnergyAndDirectionsInFlight(
          energy, dirPrimary, &theGamma1Ekin, theGamma1Dir, &theGamma2Ekin, theGamma2Dir, &rnge);

      auto &&gamma1 = secondaries.gammas.NextTrack();
      auto &&gamma2 = secondaries.gammas.NextTrack();
      atomicAdd(&globalScoring->numGammas, 2);

      InitAsSecondary(gamma1, /*parent=*/currentTrack);
      storeRng(gamma1(RngState{}), newRNG);
      gamma1(Energy{}) = theGamma1Ekin;
      gamma1(Dir{}).Set(theGamma1Dir[0], theGamma1Dir[1], theGamma1Dir[2]);

      InitAsSecondary(gamma2, /*parent=*/currentTrack);
      // Reuse the RNG state of the dying track.
      gamma2(RngState{}) = currentTrack(RngState{});
      gamma2(Energy{})   = theGamma2Ekin;
      gamma2(Dir{}).Set(theGamma2Dir[0], theGamma2Dir[1], theGamma2Dir[2]);

      // The current track is killed by not enqueuing into the next activeQueue.
      break;
    }
    }
  }
}

// Instantiate kernels for electrons and positrons.
__global__ void TransportElectrons(View electrons, const adept::MParray *active, Secondaries secondaries,
                                   adept::MParray *activeQueue, GlobalScoring *globalScoring,
                                   ScoringPerVolume *scoringPerVolume)
{
  TransportElectrons</*IsElectron*/ true>(electrons, active, secondaries, activeQueue, globalScoring, scoringPerVolume);
}
__global__ void TransportPositrons(View positrons, const adept::MParray *active, Secondaries secondaries,
                                   adept::MParray *activeQueue, GlobalScoring *globalScoring,
                                   ScoringPerVolume *scoringPerVolume)
{
  TransportElectrons</*IsElectron*/ false>(positrons, active, secondaries, activeQueue, globalScoring,
                                           scoringPerVolume);
}
